#include "hip/hip_runtime.h"
//This Kernel will be dedicated to forward feeding and backward propigation which gets error and new weights for one training pair.
// The input will be...	void training(T trainData[64][numIn_],int trueOut[64][numOut_],const int numTrainSample,
                      //const float learnRate,const long maxNumTrainIterate,float (*pLogisticFun)(float))

//The output will be the updated weights for each training pair



//This Kernel will be dedicated to updating the weights and biases for one batch

#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""

#define TILE_DIM 4                     // Tile dimension
#define DIMX 1                            
#define DIMY 2
#define DIMZ 3

__device__ float fxGPU(float *x, int idx);
void printArray(float *arr, int rows, int cols, int shouldPrint);

// a (m * n)
// b (n * k)
// c (m * k)

__global__ void matrix_multiply_simple(float* a, float* b, float* ab, int m, int n, int k) {

    int Row = blockIdx.y*blockDim.y+threadIdx.y;

    int Col = blockIdx.x*blockDim.x+threadIdx.x;

    if ((Row > m) || (Col > k)) return;

    float Pvalue = 0;
    for (int i = 0; i < (TILE_DIM + n - 1)/TILE_DIM; i++) {

        for (int p = 0; p < TILE_DIM; ++p) {
            if ((i*TILE_DIM + p < n && Row < m) && (i*TILE_DIM + p < n && Col < k))
                Pvalue += a[Row*n + i*TILE_DIM + p] * b[(i*TILE_DIM + p)*k + Col];
        }
    }
    ab[(Row*k)+Col]=Pvalue;
}

__global__ void kernel( int *input, float *output, float *vHidden, float *wHidden, float *vOut, float *wOut, float *hError, float *yError, int numIn, int numH, int numOut, int numLayers, int numPairs )
{// Done
    // need 2D indexing for input and 3D for wHidden
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*gridDim.x + ix;
    // if(ix > numTrainSample) return;

    printf("Block: %d | Thread: %d | ix: %d\n", blockIdx.x, threadIdx.x, idx);
    // for (int q=0; q<numTrainSample*numIn;q++)
    //     printf("%5d ", input[q]);
    // printf("\n");
    int h_offset = 0 + idx;
    int y_offset = numPairs + idx;
    int temp_offset = 2*numPairs + idx;
    extern __shared__ float sums[];
    int i,j,k;
    int cols = numIn + 1;
    int rows = numH;

    sums[h_offset] = 0;
    for (k=0; k<numLayers; k++) { //2x z-dim
		for(i=0; i<rows; i++){ //3x rows
			for(j=0; j<numIn; j++){ //2x, for each w1 w2 cols
                // printf("||?%5d *%5.02f||\n", input[idx*numIn+j], wHidden[k*cols*rows + i*cols + (j+1)]);
                sums[h_offset] = sums[h_offset] + input[idx*numIn+j] * wHidden[k*cols*rows + i*cols + (j+1)];
            }
            // adding the bias weight w0
            sums[h_offset] = sums[h_offset] + wHidden[k*cols*rows + i*cols + 0];
            vHidden[idx*numH+i] = fxGPU(sums, h_offset);
            // printf("%5.02f ", sums[idx]);
            sums[h_offset] = 0;
        }
        sums[h_offset] = 0;
    }

    // y (y = second half of sums) 0-63 is for sums, 64-127 is for y
    sums[y_offset] = 0;
    rows = numOut;
    int numLongLayers = 1;
    // Compute vOut
    for (k=0; k<numLongLayers; k++) { //1x z-dim
		for(i=0; i<rows; i++){ //1x for numout
            for(j=0; j<numH; j++){ //3x, for each w1 w2 w3 cols (3hidden)
                printf("||?%5d *%5.02f||\n", vHidden[idx*numH+j], wOut[k*cols*rows + i*cols + (j+1)]);
                sums[y_offset] = sums[y_offset] + vHidden[idx*numH+j] * wOut[k*cols*rows + i*cols + (j+1)];
            }
            // adding the bias weight w0
            sums[y_offset] = sums[y_offset] + wOut[k*cols*rows + i*cols + 0];
            vOut[idx*rows+i] = fxGPU(sums, y_offset);
            printf("%5.02f ", sums[y_offset]);
            sums[y_offset] = 0;
        }
    }

    // compute yErr
    for(i = 0; i < numOut; i++) {
        yError[idx*rows+i] =  vOut_[idx*numOut+i] * ( 1 - vOut_[idx*numOut+i]) * (  vOut_[idx*numOut+i] - output[idx*numOut+i] );
    }
    for(i = 0; i < numH; i++) {
        hError[idx*numH+i] = 1.2;
    }
    //compute hError
    // for(m = 0; m < numNeuronHidden_; m++)
    // {
    //     temp = 0;
    //     for(k = 0; k < numNeuronOut_; k ++)
    //         temp = temp + wOut_[k][m + 1] * yError[k];
    //     hError[m] = temp * vHidden_[m] * (1 - vHidden_[m]);

    // }
    // for(m = 0; m < numOut; m++)
    // {
    //     for(k = 0; k < numNeuronHidden_; k++)
    //         y = y + vHidden[k] * wOut_[m][k + 1];
    //     y = y + wOut_[m][0];
    //     vOut_[m] = pLogisticFun(static_cast<float>(y));

    //     y = 0;
    // }


    				// printf("%5.02f ", arr[k*cols*rows + i*cols + j]);
			// }
    // for(int layer=0; layer < numLayers; layer++) {
    //     for(int m = 0; m < numH_; m++) {
    //         for(int k = 0; k < numIn; k++) {
    //             i*cols + j
    //             atomicAdd(&sums, input[k*numIn] * wHidden[m][k + 1]);
    // atomicAdd(&sums[0], wHidden[k*cols*rows + i*cols + 0]);

    // // compute vOut
    // for(int m = 0; m < numNeuronOut_; m++)
    // {
    //     for(k = 0; k < numNeuronHidden_; k++)
    //         y = y + vHidden_[k] * wOut_[m][k + 1];
    //     y = y + wOut_[m][0];
    //     vOut_[m] = pLogisticFun(static_cast<float>(y));

    //     y = 0;
    // }
    
}

// if (ix == 0)
//         wHidden[idx] = 1;
//     else
//         wHidden[idx] = 2;

// // Indexing into wHideen, set bias=1
// int h_idx = iy*(numIn + 1) + ix;
//     if (ix == 0)
//         wHidden[h_idx] = 1;
//     else
//         wHidden[h_idx] = 2;

//the transfer function used by neural network
__device__ float fxGPU(float *x, int idx)
{
	// return (float)(1.0f / (1 + exp((float)(x * (-1)))));
    return 1.0 * x[idx];
}

void printArray(float *arr, int rows, int cols, int shouldPrint){
    if (!shouldPrint)
       return;
           
    int i,j;
 
    for(i=0; i<rows; i++){
       for(j=0; j<cols; j++){
       
          printf("%5.02f ", arr[i*cols + j]);
       }
       printf("\n");
    }
 
    printf("\n");
 }

 void printArray(int *arr, int rows, int cols, int shouldPrint){
    if (!shouldPrint)
       return;
           
    int i,j;
 
    for(i=0; i<rows; i++){
       for(j=0; j<cols; j++){
       
          printf("%d ", arr[i*cols + j]);
       }
       printf("\n");
    }
 
    printf("\n");
 }

void printArray3D(float *arr, int rows, int cols, int pages, int sP) {
	if (!sP)
	return;
		
 	int i,j,k;

	for (k=0; k<pages; k++) {
		printf("Layer %d\n", k);
		for(i=0; i<rows; i++){
			for(j=0; j<cols; j++){
		
				printf("%5.02f ", arr[k*cols*rows + i*cols + j]);
			}
			printf("\n");
   		}
   		printf("\n");
  	}

 printf("\n");
}

// __device__ void add(float *sums, float *other) {
//     atomicAdd(&sums, *other);
//   }