//This Kernel will be dedicated to forward feeding and backward propigation which gets error and new weights for one training pair.
// The input will be...	void training(T trainData[64][numIn_],int trueOut[64][numOut_],const int numTrainSample,
                      //const float learnRate,const long maxNumTrainIterate,float (*pLogisticFun)(float))

//The output will be the updated weights for each training pair



//This Kernel will be dedicated to updating the weights and biases for one batch

#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"


#define TILE_DIM 4                     // Tile dimension
#define DIMX 1                            
#define DIMY 2
#define DIMZ 3

void printArray(float *arr, int rows, int cols, int shouldPrint);

__global__ void MatMulNoShared(float* A, float* B, float* C, int ARows, int ACols, int BRows, int BCols, int CRows, int CCols) {

    float CValue = 0;

    int Row = blockIdx.y*TILE_DIM + threadIdx.y;
    int Col = blockIdx.x*TILE_DIM + threadIdx.x;

    for (int k = 0; k < (TILE_DIM + ACols - 1)/TILE_DIM; k++) {

        for (int n = 0; n < TILE_DIM; ++n) 
            if ((k*TILE_DIM + n < ACols && Row < ARows) && (k*TILE_DIM + n < BRows && Col < BCols))
                CValue += A[Row*ACols + k*TILE_DIM + n] * B[(k*TILE_DIM + n)*BCols + Col];

    }

    if (Row < CRows && Col < CCols) C[((blockIdx.y * blockDim.y + threadIdx.y)*CCols)+(blockIdx.x*blockDim.x)+threadIdx.x]=CValue;
}

int main() {

    int CCols = DIMZ, CRows=DIMX, ACols=DIMY, ARows=DIMX, BCols=DIMZ, BRows=DIMY;

    dim3 dimBlock(TILE_DIM, TILE_DIM, 1);
    dim3 dimGrid;

    dimGrid.x = (CCols + dimBlock.x - 1)/dimBlock.x;
    dimGrid.y = (CRows + dimBlock.y - 1)/dimBlock.y;

    float *deviceA, *deviceB, *deviceC;

    float* hostA    = (float*)malloc(DIMX*DIMY*sizeof(float));
    float* hostB    = (float*)malloc(DIMY*DIMZ*sizeof(float));
    float* hostC    = (float*)malloc(DIMX*DIMZ*sizeof(float));
    float* hostCp   = (float*)malloc(DIMX*DIMZ*sizeof(float));

    // for (int x = 0; x<DIMX; x++)
    //     for (int y = 0; y<DIMY; y++) {
    //         hostA[x*DIMY+y] = rand()/(float)RAND_MAX;
    //         hostB[x*DIMY+y] = rand()/(float)RAND_MAX;
    //     }

    hostA[0] = 1.0;
    hostA[1] = 2.0;
    hostB[0] = 1.0;
    hostB[1] = 2.0;
    hostB[3] = 3.0;
    hostB[4] = 1.0;
    hostB[5] = 2.0;
    hostB[6] = 3.0;

    hipMalloc((void **)&deviceA, DIMX*DIMY*sizeof(float));
    hipMalloc((void **)&deviceB, DIMY*DIMZ*sizeof(float));
    hipMalloc((void **)&deviceC, DIMX*DIMZ*sizeof(float));

    hipMemcpy(deviceA, hostA, DIMX*DIMY*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, DIMY*DIMZ*sizeof(float), hipMemcpyHostToDevice);

    MatMulNoShared<<<dimGrid , dimBlock>>>(deviceA , deviceB , deviceC , ARows , ACols, BRows ,BCols , CRows , CCols);

    hipMemcpy(hostC, deviceC, DIMX*DIMZ*sizeof(float), hipMemcpyDeviceToHost);
    printArray(hostA, DIMX, DIMY, 1);
    printArray(hostB, DIMY, DIMZ, 1);
    printArray(hostC, DIMX, DIMZ, 1);

    return 0;
}


void printArray(float *arr, int rows, int cols, int shouldPrint){
    if (!shouldPrint)
       return;
           
    int i,j;
 
    for(i=0; i<rows; i++){
       for(j=0; j<cols; j++){
       
          printf("%d ", arr[i*cols + j]);
       }
       printf("\n");
    }
 
    printf("\n");
 }