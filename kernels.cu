//This Kernel will be dedicated to forward feeding and backward propigation which gets error and new weights for one training pair.
// The input will be...	void training(T trainData[64][numIn_],int trueOut[64][numOut_],const int numTrainSample,
                      //const float learnRate,const long maxNumTrainIterate,float (*pLogisticFun)(float))

//The output will be the updated weights for each training pair



//This Kernel will be dedicated to updating the weights and biases for one batch

#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"


#define TILE_DIM 4                     // Tile dimension
#define DIMX 1                            
#define DIMY 2
#define DIMZ 3

__device__ float fxGPU(float *x, int idx);
void printArray(float *arr, int rows, int cols, int shouldPrint);

// a (m * n)
// b (n * k)
// c (m * k)

__global__ void matrix_multiply_simple(float* a, float* b, float* ab, int m, int n, int k) {

    int Row = blockIdx.y*blockDim.y+threadIdx.y;

    int Col = blockIdx.x*blockDim.x+threadIdx.x;

    if ((Row > m) || (Col > k)) return;

    float Pvalue = 0;
    for (int i = 0; i < (TILE_DIM + n - 1)/TILE_DIM; i++) {

        for (int p = 0; p < TILE_DIM; ++p) {
            if ((i*TILE_DIM + p < n && Row < m) && (i*TILE_DIM + p < n && Col < k))
                Pvalue += a[Row*n + i*TILE_DIM + p] * b[(i*TILE_DIM + p)*k + Col];
        }
    }
    ab[(Row*k)+Col]=Pvalue;
}

__global__ void kernel( int *input, float *output, float *vHidden, float *wHidden, float *vOut, float *wOut, float *hError, float *yError, int numIn, int numH, int numOut, int numLayers, int numPairs )
{// Done
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*gridDim.x + ix;
    // if(ix > numTrainSample) return;

    printf("Block: %d | Thread: %d | ix: %d\n", blockIdx.x, threadIdx.x, idx);
    // for (int q=0; q<numTrainSample*numIn;q++)
    //     printf("%5d ", input[q]);
    // printf("\n");
    int h_offset = 0 + idx;
    int y_offset = numPairs + idx;
    int temp_offset = 2*numPairs + idx;
    extern __shared__ float sums[];
    int i,j,k;
    int cols = numIn + 1;
    int rows = numH;

    sums[h_offset] = 0;
    for (k=0; k<numLayers; k++) { //2x z-dim
		for(i=0; i<rows; i++){ //3x rows
			for(j=0; j<numIn; j++){ //2x, for each w1 w2 cols
                // printf("||?%5d *%5.02f||\n", input[idx*numIn+j], wHidden[k*cols*rows + i*cols + (j+1)]);
                sums[h_offset] = sums[h_offset] + input[idx*numIn+j] * wHidden[k*cols*rows + i*cols + (j+1)];
            }
            // adding the bias weight w0
            sums[h_offset] = sums[h_offset] + wHidden[k*cols*rows + i*cols + 0];
            vHidden[idx*numH+i] = fxGPU(sums, h_offset);
            // printf("%5.02f ", sums[idx]);
            sums[h_offset] = 0;
        }
        sums[h_offset] = 0;
    }

    // y (y = second half of sums) 0-63 is for h, 64-127 is for y, 128-... is for temp
    sums[y_offset] = 0;
    rows = numOut;
    cols = numH;
    int numLongLayers = 1;
    // Compute vOut
    for (k=0; k<numLongLayers; k++) { //1x z-dim
		for(i=0; i<rows; i++){ //1x for numout
            for(j=0; j<numH; j++){ //3x, for each w1 w2 w3 cols (3hidden)
                // printf("||?%5d *%5.02f||\n", vHidden[idx*numH+j], wOut[k*cols*rows + i*cols + (j+1)]);
                sums[y_offset] = sums[y_offset] + vHidden[idx*numH+j] * wOut[k*cols*rows + i*cols + (j+1)];
            }
            // adding the bias weight w0
            sums[y_offset] = sums[y_offset] + wOut[k*cols*rows + i*cols + 0];
            vOut[idx*rows+i] = fxGPU(sums, y_offset);
            printf("%5.02f ", sums[y_offset]);
            sums[y_offset] = 0;
        }
    }
    // // compute yErr
    // for(i = 0; i < numOut; i++) {
    //     yError[idx*numOut+i] =  vOut[idx*numOut+i] * ( 1 - vOut[idx*numOut+i]) * (  vOut[idx*numOut+i] - output[idx*numOut+i] );
    // }
    // sums[temp_offset] = 0;
    // // compute hErr
    // for (k=0; k<numLongLayers; k++) { //for future z dim is num layers
    //     for(j = 0; j < numH; j++) { // j is for cols, numH
    //         sums[temp_offset] = 0;
    //         for(i = 0; i < numOut; i++) { // i is for rows, 1x for numOut
    //             // wOut -> [wbias, w1, w2, w3]xnumOut, doing [w1-w3] now
    //             sums[temp_offset] = sums[temp_offset] + wOut[k*cols*rows + i*cols + (j+1)] * yError[idx*numOut+i];
    //             // yError[idx*numOut+i] =  vOut[idx*numOut+i] * ( 1 - vOut[idx*numOut+i]) * (  vOut[idx*numOut+i] - output[idx*numOut+i] );
    //         }
    //         printf("vHidden: %f | wOut: %f | yErr: %f\n", vHidden[idx*numH+j], wOut[k*cols*rows + i*cols + (j+1)], yError[idx*numOut+i]);
    //         hError[idx*numH+j] = sums[temp_offset] * vHidden[idx*numH+j]*(1 - vHidden[idx*numH+j]);
    //     }
    // }
    // for(m = 0; m < numNeuronOut_; m++)
    //                 yError[m] =  vOut_[m] * ( 1 - vOut_[m]) * (  vOut_[m] - trueOut[i][m] );
    //compute hError
    // for(m = 0; m < numNeuronHidden_; m++)
    // {
    //     temp = 0;
    //     for(k = 0; k < numNeuronOut_; k ++)
    //         temp = temp + wOut_[k][m + 1] * yError[k];
    //     hError[m] = temp * vHidden_[m] * (1 - vHidden_[m]);

    // }
    // for(m = 0; m < numOut; m++)
    // {
    //     for(k = 0; k < numNeuronHidden_; k++)
    //         y = y + vHidden[k] * wOut_[m][k + 1];
    //     y = y + wOut_[m][0];
    //     vOut_[m] = pLogisticFun(static_cast<float>(y));

    //     y = 0;
    // }


    				// printf("%5.02f ", arr[k*cols*rows + i*cols + j]);
			// }
    // for(int layer=0; layer < numLayers; layer++) {
    //     for(int m = 0; m < numH_; m++) {
    //         for(int k = 0; k < numIn; k++) {
    //             i*cols + j
    //             atomicAdd(&sums, input[k*numIn] * wHidden[m][k + 1]);
    // atomicAdd(&sums[0], wHidden[k*cols*rows + i*cols + 0]);

    // // compute vOut
    // for(int m = 0; m < numNeuronOut_; m++)
    // {
    //     for(k = 0; k < numNeuronHidden_; k++)
    //         y = y + vHidden_[k] * wOut_[m][k + 1];
    //     y = y + wOut_[m][0];
    //     vOut_[m] = pLogisticFun(static_cast<float>(y));

    //     y = 0;
    // }
    
}

__global__ void adjustWeights(float learnRate, float *wHidden, float *wOut, float *hError, float *yError, int numIn, int numH, int numOut, int numLayers, int numPairs )
{// Done
    // need 2D indexing for input and 3D for wHidden
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*gridDim.x + ix;
    // if(ix > numTrainSample) return;


    //Adjust wOut[i][0] and wOut[i][j] and wHidden_
    // adjust bias weight for wOut
    // for(m = 0; m < numNeuronOut_; m++)
    //     wOut_[m][0] = wOut_[m][0] - learnRate * yError[m];

    // // adjust wOut general weights
    // for(m = 0; m < numNeuronOut_; m++)
    //     for(k = 0; k < numNeuronHidden_; k++)
    //         wOut_[m][k + 1] = wOut_[m][k + 1] - learnRate * yError[m] * vHidden_[k];

    // // adjust bias weight for wHidden (outer) and wHidden weights (inner)
    // for(m = 0; m < numNeuronHidden_; m++)
    // {
    //     wHidden_[m][0] = wHidden_[m][0] - learnRate * hError[m];
    //     for(k = 0; k < numNeuronIn_; k++)
    //         wHidden_[m][k + 1] = wHidden_[m][k + 1] - learnRate * hError[m] * indata_[k];
    // }
}
// if (ix == 0)
//         wHidden[idx] = 1;
//     else
//         wHidden[idx] = 2;

// // Indexing into wHideen, set bias=1
// int h_idx = iy*(numIn + 1) + ix;
//     if (ix == 0)
//         wHidden[h_idx] = 1;
//     else
//         wHidden[h_idx] = 2;

//the transfer function used by neural network
__device__ float fxGPU(float *x, int idx)
{
	// return (float)(1.0f / (1 + exp((float)(x * (-1)))));
    return 1.0 * x[idx];
}

void printArray(float *arr, int rows, int cols, int shouldPrint){
    if (!shouldPrint)
       return;
           
    int i,j;
 
    for(i=0; i<rows; i++){
       for(j=0; j<cols; j++){
       
          printf("%5.02f ", arr[i*cols + j]);
       }
       printf("\n");
    }
 
    printf("\n");
 }

 void printArray(int *arr, int rows, int cols, int shouldPrint){
    if (!shouldPrint)
       return;
           
    int i,j;
 
    for(i=0; i<rows; i++){
       for(j=0; j<cols; j++){
       
          printf("%d ", arr[i*cols + j]);
       }
       printf("\n");
    }
 
    printf("\n");
 }

void printArray3D(float *arr, int rows, int cols, int pages, int sP) {
	if (!sP)
	return;
		
 	int i,j,k;

	for (k=0; k<pages; k++) {
		printf("Layer %d\n", k);
		for(i=0; i<rows; i++){
			for(j=0; j<cols; j++){
		
				printf("%5.02f ", arr[k*cols*rows + i*cols + j]);
			}
			printf("\n");
   		}
   		printf("\n");
  	}

 printf("\n");
}

// __device__ void add(float *sums, float *other) {
//     atomicAdd(&sums, *other);
//   }