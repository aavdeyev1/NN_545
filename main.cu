#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;
#include <cstdlib>
#include <ctime>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include <fstream>
#include <cstring>
#include <iomanip>
#include <limits>
#include <string.h>

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include "cpu.h"
#include "kernels.h"

// Make static array for these numbers so we can grow the number of hidden layers
#define numIn_ 2
#define numH_ 3
#define numOut_ 1
#define numTLayers 1 // HIDDEN LAYER W/O OUTPUT LAYER

// #define numTrainSample_ 64
#define numTrainSample_ 4

void training(int *trainData, int *trueOut, const int numTrainSample,const float learnRate,const long maxNumTrainIterate);

int main()
{
	int x = 2, y=2;
	int indata[2][2] = {
		{ 1,1},
		{ 1,0}
	};
    // int indata[y][x] = {
    //                         { 1,1,1,1, 1,1,1,1},
    //                         { 1,1,1,1, 1,1,1,1},
    //                         { 1,1,0,1, 1,1,1,1},
    //                         { 1,0,0,0, 1,1,1,1},
    //                         { 1,0,0,0, 0,0,1,1},
    //                         { 0,0,0,0, 0,0,1,1},
    //                         { 0,0,0,0, 0,1,1,1},
    //                         { 0,0,0,1, 1,1,1,1}
    //                     };
    __device__ float fxGPU(float); // init activation fn

    // Need linearized input/output for GPU.
    int i,j,k = 0,input[numIn_*numTrainSample_] = {0},output[numOut_*numTrainSample_] = {0};

    for(i = 0; i < x; i ++)
        for(j = 0; j< y; j++)
        {
            // k = rowNum
            input[k*numIn_] = i;
            input[k*numIn_ + 1] = j;
            output[k] = indata[i][j];
            k ++;
        }

    printArray(input, numTrainSample_, numIn_, 1);
    printArray(output, 1, numTrainSample_, 1);

    training(input, output, numTrainSample_,0.02f,1l);


    // bpNeuralNetwork<int> myBPNN;
    // myBPNN.training( input,output,64,0.02f,100000l,fx);
    cout << "\n\n\n                Press any key to exit!";
    getchar();
    return 0;
}


// Make hipMemcpy and hipMalloc to allocate memory for gpu
// Input will be the input and output arrays calculated in main numTrainSample, learnRate, long maxNumTrainIterate, *pLogisticFun
void training(int *trainData, int *trueOut, const int numTrainSample,const float learnRate,const long maxNumTrainIterate)
{
    // row number of the trainData is the amounts of training samples, the column of the trainData  that is from column 0 to numNeuronIn_ - 1 will
		// be assigned to indata_ .
		// pointer of pLogisticFun, is a function pointer, that enable us to use other logistic function in training conveniently
		// number of rows of trueOut is equal to trainData's row number;One trueOut row corresponds to one trainData row. 
		long iterate = 0L;
		int i,k,m;
		float h = 0;
		float y = 0;
		float temp = 0;
		float* yError;
		float* hError;
		int numE = 0;
		int width = 6;

		float mytrim(float);

        // int* d_indata;
		// float* d_vHidden;
		// float* h_wHidden;
		float* d_h;
		// float* d_vOut;
		// float* d_yError;
		// float* d_hError;
		// float* d_wOut;
		float* d_result;

		
		
		//setup block and grid size
		int blockSize, gridSize;
		blockSize = 1;
		gridSize = 1;

        float* testW = (float *)malloc(numTLayers*numH_*(numIn_ + 1)*sizeof(float));
		float* h_W = (float *)malloc(numTLayers*numH_*(numIn_ + 1)*sizeof(float));
		testW[0] = 0.0;
		testW[1] = 1.0;
		testW[2] = 2.0;
		testW[3] = 3.0;
		testW[4] = 4.0;
		testW[5] = 5.0;
		testW[6] = 6.0;
		testW[7] = 7.0;
		testW[8] = 8.0;

		float *wOutTestIn = (float *)malloc(numOut_*(numH_+1)*sizeof(float));
		wOutTestIn[0] = 0.0;
		wOutTestIn[1] = 1.0;
		wOutTestIn[2] = 2.0;
		wOutTestIn[3] = 3.0;
		// testW[9] = 91.0;
		// testW[10] = 92.0;
		// testW[11] = 93.0;
		// testW[12] = 94.0;
		// testW[13] = 95.0;
		// testW[14] = 96.0;
		// testW[15] = 97.0;
		// testW[16] = 98.0;
		// testW[17] = 99.0;
		printf("wHidden:\n");
		printArray3D(testW, numH_, numIn_ + 1, numTLayers, 1);
		printf("wOut:\n");
		printArray3D(wOutTestIn, numOut_, numH_ + 1, 1, 1);

		// printArray3D(testW, , 4, 1, 1); // cols, rows, 

        // Allocate host mem
        int *h_input=0;
        float *h_output=0;
		float *h_vHidden=0;
		float *h_wHidden=0;
		float *h_vOut=0;
		float *h_wOut=0;
		float *h_yError=0;
		float *h_hError=0;
        h_input = (int *)malloc(numIn_*numTrainSample_*sizeof(int));
        h_output = (float *)malloc(numOut_*numTrainSample_*sizeof(float));
        h_vHidden = (float *)malloc(numTrainSample_*numH_*sizeof(float));
		h_wHidden = (float *)malloc(numTLayers*numH_*(numIn_+1)*sizeof(float)); // 3D by Layer, numNeuron, numWeight
		h_vOut = (float *)malloc(numOut_*numTrainSample_*sizeof(float));
		h_wOut = (float *)malloc(numOut_*(numH_+1)*sizeof(float)); // 3D by Layer, numNeuron, numWeight
		h_yError = (float *)malloc(numOut_*numTrainSample_*sizeof(float));
		h_hError = (float *)malloc(numTrainSample_*numH_*sizeof(float));
		
		// d_h = (float *)malloc(); // TBD
		// float* d_vOut;
		// float* d_yError;
		// float* d_hError;
		// float* d_wOut;
		// float* d_result;
        // error vector

        // Allocate dev mem
        int *d_input=0;
		float *d_output=0;
		float *d_vHidden=0;
		float *d_wHidden=0;
		float *d_vOut=0;
		float *d_wOut=0;
		float *d_yError=0;
		float *d_hError=0;
        checkCudaErrors( hipMalloc( &d_input, numIn_*numTrainSample_*sizeof(int) ) );
        checkCudaErrors( hipMalloc( &d_output, numOut_*numTrainSample_*sizeof(float) ) );
        checkCudaErrors( hipMalloc( &d_vHidden, numTrainSample_*numH_*sizeof(float) ) );
        checkCudaErrors( hipMalloc( &d_wHidden, numTLayers*numH_*(numIn_+1)*sizeof(float) ) );
        checkCudaErrors( hipMalloc( &d_vOut, numOut_*numTrainSample_*sizeof(float) ) );
        checkCudaErrors( hipMalloc( &d_wOut, numOut_*(numH_+1)*sizeof(float) ) );
		checkCudaErrors( hipMalloc( &d_yError, numOut_*numTrainSample_*sizeof(float) ) );
        checkCudaErrors( hipMalloc( &d_hError, numTrainSample_*numH_*sizeof(float) ) );

        checkCudaErrors( hipMemcpy( d_input, trainData, numIn_*numTrainSample_*sizeof(int), hipMemcpyHostToDevice) );
        checkCudaErrors( hipMemcpy( d_output, trueOut, numOut_*numTrainSample_*sizeof(float), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy( d_wHidden, testW, numTLayers*numH_*(numIn_+1)*sizeof(float), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy( d_wOut, wOutTestIn, numOut_*(numH_+1)*sizeof(float), hipMemcpyHostToDevice) );


        dim3 grid, block;

        block.x = 4;
        grid.x  = ceil( (float)numTrainSample_ / block.x );
        // grid.y  = ceil( (float)numTrainSample_ / block.y );
        
        kernel<<<grid, block, 3*numTrainSample_*sizeof(float)>>>(d_input,
								 d_output,
								 d_vHidden,
								 d_wHidden,
								 d_vOut,
								 d_wOut,
								 d_hError,
								 d_yError,
								 numIn_,
								 numH_,
								 numOut_,
								 numTLayers,
								 numTrainSample_);
        
		hipDeviceSynchronize();
		hipError_t err = hipGetLastError();        // Get error code

		if ( err != hipSuccess )
		{
		printf("CUDA Error: %s\n", hipGetErrorString(err));
		exit(-1);
		}
		checkCudaErrors( hipMemcpy( h_input, d_input, numIn_*numTrainSample_*sizeof(int), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy( h_output, d_output, numOut_*numTrainSample_*sizeof(float), hipMemcpyDeviceToHost ) );

        checkCudaErrors( hipMemcpy( h_W, d_wHidden, numTLayers*numH_*(numIn_ + 1)*sizeof(float), hipMemcpyDeviceToHost ) );
		checkCudaErrors( hipMemcpy( h_vHidden, d_hError, numTrainSample_*numH_*sizeof(float), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy( h_vOut, d_yError, numOut_*numTrainSample_*sizeof(float), hipMemcpyDeviceToHost ) );
		checkCudaErrors( hipMemcpy( h_wOut, d_wOut, numOut_*(numH_+1)*sizeof(float), hipMemcpyDeviceToHost ) );
        
		printf("Input:\n");
        printArray(h_input, numTrainSample_, numIn_, 1);
        // printArray(h_output, 1, numTrainSample_, 1);
		printf("hidden weights:\n");
		printArray3D(h_W, numH_, numIn_+1, numTLayers, 1);

		printf("vHidden HERROR:\n");
		printArray(h_vHidden, numTrainSample_, numH_, 1);

		printf("out weights YERROR:\n");
		printArray3D(h_wOut, numOut_, numH_+1, 1, 1);

		printf("vOut:\n");
		printArray(h_vOut, numTrainSample_, numOut_, 1);

        free( h_input );
        free( h_output );
		free( h_vHidden );
		free(h_wHidden);
		free(h_vOut);
		free(h_wOut);

        free(testW);
		free(h_W);
		free(wOutTestIn);
        
        hipFree( d_input );
        hipFree( d_output );
        hipFree( d_vHidden );
		hipFree( d_wHidden );
		hipFree(d_vOut);
		hipFree(d_wOut);
        
    // for each training interation in maxNumTrainIterations

    // update weights KERNEL!

    // for each training pair k:




        // kernel call: 1 per input
        // inputs: trainDataInput vector
        //  trueOut vector, length numOut
        //  hidden weights H

        

		
	// 	for(iterate = 1; iterate <= maxNumTrainIterate; iterate ++)
	// 	{
	// 		for(i = 0; i < numTrainSample; i++)
	// 		{
	// 			// make an input vector of len num of input data, for a given training sample i
	// 			for(k = 0; k < numIn_; k++)
	// 				indata_[k] = trainData[i][k];
				

	// 			// forward computing
	// 			//
	// 			//
	// 			// compute vHidden
	// 			for(m = 0; m < numNeuronHidden_; m++)
	// 			{
	// 				for(k = 0; k < numNeuronIn_; k++)
	// 					h = h + indata_[k] * wHidden_[m][k + 1];
	// 				h = h + wHidden_[m][0];
	// 				vHidden_[m] = pLogisticFun(static_cast<float>(h));

	// 				h = 0;
	// 			}

	// 			// compute vOut
	// 			for(m = 0; m < numNeuronOut_; m++)
	// 			{
	// 				for(k = 0; k < numNeuronHidden_; k++)
	// 					y = y + vHidden_[k] * wOut_[m][k + 1];
	// 				y = y + wOut_[m][0];
	// 				vOut_[m] = pLogisticFun(static_cast<float>(y));

	// 				y = 0;
	// 			}

	// 			//
	// 			//
	// 			//backward compute

	// 			//compute yError
	// 			for(m = 0; m < numNeuronOut_; m++)
	// 				yError[m] =  vOut_[m] * ( 1 - vOut_[m]) * (  vOut_[m] - trueOut[i][m] );
				
	// 			//compute hError
	// 			for(m = 0; m < numNeuronHidden_; m++)
	// 			{
	// 				temp = 0;
	// 				for(k = 0; k < numNeuronOut_; k ++)
	// 					temp = temp + wOut_[k][m + 1] * yError[k];
	// 				hError[m] = temp * vHidden_[m] * (1 - vHidden_[m]);

	// 			}

	// 			//Adjust wOut[i][0] and wOut[i][j] and wHidden_
	// 			for(m = 0; m < numNeuronOut_; m++)
	// 				wOut_[m][0] = wOut_[m][0] - learnRate * yError[m];

	// 			for(m = 0; m < numNeuronOut_; m++)
	// 				for(k = 0; k < numNeuronHidden_; k++)
    //                     wOut_[m][k + 1] = wOut_[m][k + 1] - learnRate * yError[m] * vHidden_[k];

	// 			for(m = 0; m < numNeuronHidden_; m++)
	// 			{
	// 				wHidden_[m][0] = wHidden_[m][0] - learnRate * hError[m];
	// 				for(k = 0; k < numNeuronIn_; k++)
	// 					wHidden_[m][k + 1] = wHidden_[m][k + 1] - learnRate * hError[m] * indata_[k];
	// 			}
				
	// 			//one statement below did not consider the general neural network constructure, just for this assignment
	// 			result[static_cast<int>(indata_[0])][static_cast<int>(indata_[1])] = vOut_[0];
			
	// 		}// end for all samples

	// 		} // 

	// 	} // end for iteration
		
	// }// end for training


}